#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <>

#include "safecall.hpp"

#define X 0
#define Y 1
#define Z 2

#define CROSS(dest,v1,v2) \
          dest[0]=v1[1]*v2[2]-v1[2]*v2[1]; \
          dest[1]=v1[2]*v2[0]-v1[0]*v2[2]; \
          dest[2]=v1[0]*v2[1]-v1[1]*v2[0]; 

#define DOT(v1,v2) (v1[0]*v2[0]+v1[1]*v2[1]+v1[2]*v2[2])

#define SUB(dest,v1,v2) \
          dest[0]=v1[0]-v2[0]; \
          dest[1]=v1[1]-v2[1]; \
          dest[2]=v1[2]-v2[2]; 

#define MAX(x, y) x > y? x:y
#define MIN(x, y) x < y? x:y

#define FINDMINMAX(x0,x1,x2,min,max) \
  min = max = x0;   \
  if(x1<min) min=x1;\
  if(x1>max) max=x1;\
  if(x2<min) min=x2;\
  if(x2>max) max=x2;

__device__ __forceinline__
int planeBoxOverlap(float normal[3], float vert[3], float maxbox[3])	// -NJMP-
{
    int q;
    float vmin[3], vmax[3], v;
    for (q = X; q <= Z; q++)
    {
        v = vert[q];					// -NJMP-
        if (normal[q] > 0.0f)
        {
            vmin[q] = -maxbox[q] - v;	// -NJMP-
            vmax[q] = maxbox[q] - v;	// -NJMP-
        }
        else
        {
            vmin[q] = maxbox[q] - v;	// -NJMP-
            vmax[q] = -maxbox[q] - v;	// -NJMP-
        }
    }
    if (DOT(normal, vmin) > 0.0f) return 0;	// -NJMP-
    if (DOT(normal, vmax) >= 0.0f) return 1;	// -NJMP-

    return 0;
}


/*======================== X-tests ========================*/
#define AXISTEST_X01(a, b, fa, fb)			   \
	p0 = a*v0[Y] - b*v0[Z];			       	   \
	p2 = a*v2[Y] - b*v2[Z];			       	   \
        if(p0<p2) {min=p0; max=p2;} else {min=p2; max=p0;} \
	rad = fa * boxhalfsize[Y] + fb * boxhalfsize[Z];   \
	if(min>rad || max<-rad) return 0;

#define AXISTEST_X2(a, b, fa, fb)			   \
	p0 = a*v0[Y] - b*v0[Z];			           \
	p1 = a*v1[Y] - b*v1[Z];			       	   \
        if(p0<p1) {min=p0; max=p1;} else {min=p1; max=p0;} \
	rad = fa * boxhalfsize[Y] + fb * boxhalfsize[Z];   \
	if(min>rad || max<-rad) return 0;

/*======================== Y-tests ========================*/
#define AXISTEST_Y02(a, b, fa, fb)			   \
	p0 = -a*v0[X] + b*v0[Z];		      	   \
	p2 = -a*v2[X] + b*v2[Z];	       	       	   \
        if(p0<p2) {min=p0; max=p2;} else {min=p2; max=p0;} \
	rad = fa * boxhalfsize[X] + fb * boxhalfsize[Z];   \
	if(min>rad || max<-rad) return 0;

#define AXISTEST_Y1(a, b, fa, fb)			   \
	p0 = -a*v0[X] + b*v0[Z];		      	   \
	p1 = -a*v1[X] + b*v1[Z];	     	       	   \
        if(p0<p1) {min=p0; max=p1;} else {min=p1; max=p0;} \
	rad = fa * boxhalfsize[X] + fb * boxhalfsize[Z];   \
	if(min>rad || max<-rad) return 0;

/*======================== Z-tests ========================*/

#define AXISTEST_Z12(a, b, fa, fb)			   \
	p1 = a*v1[X] - b*v1[Y];			           \
	p2 = a*v2[X] - b*v2[Y];			       	   \
        if(p2<p1) {min=p2; max=p1;} else {min=p1; max=p2;} \
	rad = fa * boxhalfsize[X] + fb * boxhalfsize[Y];   \
	if(min>rad || max<-rad) return 0;

#define AXISTEST_Z0(a, b, fa, fb)			   \
	p0 = a*v0[X] - b*v0[Y];				   \
	p1 = a*v1[X] - b*v1[Y];			           \
        if(p0<p1) {min=p0; max=p1;} else {min=p1; max=p0;} \
	rad = fa * boxhalfsize[X] + fb * boxhalfsize[Y];   \
	if(min>rad || max<-rad) return 0;

__device__ __forceinline__
int triBoxOverlap(float boxcenter[3], float boxhalfsize[3], float triverts[3][3])
{

    /*    use separating axis theorem to test overlap between triangle and box */
    /*    need to test for overlap in these directions: */
    /*    1) the {x,y,z}-directions (actually, since we use the AABB of the triangle */
    /*       we do not even need to test these) */
    /*    2) normal of the triangle */
    /*    3) crossproduct(edge from tri, {x,y,z}-directin) */
    /*       this gives 3x3=9 more tests */
    float v0[3], v1[3], v2[3];
    //   float axis[3];
    float min, max, p0, p1, p2, rad, fex, fey, fez;		// -NJMP- "d" local variable removed
    float normal[3], e0[3], e1[3], e2[3];

    /* This is the fastest branch on Sun */
    /* move everything so that the boxcenter is in (0,0,0) */
    SUB(v0, triverts[0], boxcenter);
    SUB(v1, triverts[1], boxcenter);
    SUB(v2, triverts[2], boxcenter);

    /* compute triangle edges */
    SUB(e0, v1, v0);      /* tri edge 0 */
    SUB(e1, v2, v1);      /* tri edge 1 */
    SUB(e2, v0, v2);      /* tri edge 2 */

    /* Bullet 3:  */
    /*  test the 9 tests first (this was faster) */
    fex = fabsf(e0[X]);
    fey = fabsf(e0[Y]);
    fez = fabsf(e0[Z]);
    AXISTEST_X01(e0[Z], e0[Y], fez, fey);
    AXISTEST_Y02(e0[Z], e0[X], fez, fex);
    AXISTEST_Z12(e0[Y], e0[X], fey, fex);

    fex = fabsf(e1[X]);
    fey = fabsf(e1[Y]);
    fez = fabsf(e1[Z]);
    AXISTEST_X01(e1[Z], e1[Y], fez, fey);
    AXISTEST_Y02(e1[Z], e1[X], fez, fex);
    AXISTEST_Z0(e1[Y], e1[X], fey, fex);

    fex = fabsf(e2[X]);
    fey = fabsf(e2[Y]);
    fez = fabsf(e2[Z]);
    AXISTEST_X2(e2[Z], e2[Y], fez, fey);
    AXISTEST_Y1(e2[Z], e2[X], fez, fex);
    AXISTEST_Z12(e2[Y], e2[X], fey, fex);

    /* Bullet 1: */
     /*  first test overlap in the {x,y,z}-directions */
     /*  find min, max of the triangle each direction, and test for overlap in */
     /*  that direction -- this is equivalent to testing a minimal AABB around */
     /*  the triangle against the AABB */

     /* test in X-direction */
    FINDMINMAX(v0[X], v1[X], v2[X], min, max);
    if (min > boxhalfsize[X] || max < -boxhalfsize[X]) return 0;

    /* test in Y-direction */
    FINDMINMAX(v0[Y], v1[Y], v2[Y], min, max);
    if (min > boxhalfsize[Y] || max < -boxhalfsize[Y]) return 0;

    /* test in Z-direction */
    FINDMINMAX(v0[Z], v1[Z], v2[Z], min, max);
    if (min > boxhalfsize[Z] || max < -boxhalfsize[Z]) return 0;

    /* Bullet 2: */
    /*  test if the box intersects the plane of the triangle */
    /*  compute plane equation of triangle: normal*x+d=0 */
    CROSS(normal, e0, e1);
    // -NJMP- (line removed here)
    if (!planeBoxOverlap(normal, v0, boxhalfsize)) return 0;	// -NJMP-

    return 1;   /* box and triangle overlaps */
}

__global__ void setIntersectVoxels_kernel(float3 *_vertices_ptr, int3* _faces_ptr, int* _volume, int3 _vol_res, float3 _vol_min_corner, float3 _vol_max_corner, int _vertices_num, int _face_num)
{
    float3 step;
    step.x = (_vol_max_corner.x - _vol_min_corner.x) / _vol_res.x;
    step.y = (_vol_max_corner.y - _vol_min_corner.y) / _vol_res.y;
    step.z = (_vol_max_corner.z - _vol_min_corner.z) / _vol_res.z;
    float boxhalfsize[3] = { step.x / 2.f, step.y / 2.f, step.z / 2.f };

    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if (idx < _face_num)
    {
        int3 face = _faces_ptr[idx];
        float3 v0 = _vertices_ptr[face.x];
        float3 v1 = _vertices_ptr[face.y];
        float3 v2 = _vertices_ptr[face.z];

        float triverts[3][3] = { {v0.x, v0.y, v0.z}, {v1.x, v1.y, v1.z}, {v2.x, v2.y, v2.z} };

        float x_min, x_max, y_min, y_max, z_min, z_max;
        FINDMINMAX(v0.x, v1.x, v2.x, x_min, x_max);
        FINDMINMAX(v0.y, v1.y, v2.y, y_min, y_max);
        FINDMINMAX(v0.z, v1.z, v2.z, z_min, z_max);

        int3 bb_min_corner, bb_max_corner;
        bb_min_corner.x = MAX(int(floor((x_min - _vol_min_corner.x) / step.x)), 0);
        bb_min_corner.y = MAX(int(floor((y_min - _vol_min_corner.y) / step.y)), 0);
        bb_min_corner.z = MAX(int(floor((z_min - _vol_min_corner.z) / step.z)), 0);

        bb_max_corner.x = MIN(int(ceil((x_max - _vol_min_corner.x) / step.x)), _vol_res.x);
        bb_max_corner.y = MIN(int(ceil((y_max - _vol_min_corner.y) / step.y)), _vol_res.y);
        bb_max_corner.z = MIN(int(ceil((z_max - _vol_min_corner.z) / step.z)), _vol_res.z);

        for (int xx = bb_min_corner.x; xx < bb_max_corner.x; xx++)
        {
            for (int yy = bb_min_corner.y; yy < bb_max_corner.y; yy++)
            {
                for (int zz = bb_min_corner.z; zz < bb_max_corner.z; zz++)
                {
                    float boxcenter_x = xx*step.x + boxhalfsize[0]+ _vol_min_corner.x;
                    float boxcenter_y = yy*step.y + boxhalfsize[1]+ _vol_min_corner.y;
                    float boxcenter_z = zz*step.z + boxhalfsize[2]+ _vol_min_corner.z;

                    float boxcenter[3] = { boxcenter_x, boxcenter_y, boxcenter_z };
                    if (triBoxOverlap(boxcenter, boxhalfsize, triverts))
                    {
                        int id = zz*_vol_res.y*_vol_res.x + yy*_vol_res.x + xx;
                        atomicMax(_volume + id, 1);
                    }
                }
            }
        }
    }
}

void setIntersectVoxels(float3 *_vertices_ptr, int3* _faces_ptr, int* _volume, int3 _vol_res, float3 _vol_min_corner, float3 _vol_max_corner, int _vertices_num, int _face_num)
{
    dim3 block(64);
    dim3 grid((_face_num + 63) / 64);
    setIntersectVoxels_kernel << <grid, block >> > (_vertices_ptr, _faces_ptr, _volume, _vol_res, _vol_min_corner, _vol_max_corner, _vertices_num, _face_num);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}